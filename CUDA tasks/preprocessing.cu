#include "hip/hip_runtime.h"
#include "preprocessing.h"
#include "kernels.h"
#include<hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/scan.h>
#include <thrust/execution_policy.h>

// Function to create a new node
Node *createNode(int v, int weighted, int wt)
{
    Node *node = (Node *)malloc(sizeof(Node));
    node->vertex = v;
    node->next = NULL;

    if (weighted == 1)
        node->wt = wt;
    else
        wt = 1;

    return node;
}

bool comp_Edges_and_dest(Edge &a, Edge &b)
{
    return a.src == b.src ? a.dest < b.dest : a.src < b.src;
}

bool comp_Edges(Edge &a, Edge &b)
{
    return a.src < b.src;
}

int takeChoices(int& directed, int& weighted, int& algoChoice, string& filename, int& sortedOption, string& filenameforCorrection){
    filename += "../../Graphs/";

    string ext = ".mtx";
    int fileNo;

    cout << "Do you want the edge list to be in sorted order? Enter 1 for Yes or 0 for No: ";
    cin >> sortedOption;

    cout << endl;
    cout << "Choose input file: "<< endl;
    cout << " 1. chesapeake               2. rgg_n_2_16_s0             3. kron_g500-logn16         4. inf-luxembourg_osm       5. delaunay_n17" << endl;
    cout << " 6. co-papers-citeseer       7. co-papers-dblp            8. kron_g500-logn21         9. hugetrace-00000         10. channel-500x100x100-b050" << endl;
    cout << "11. delaunay_n23            12. hugetrace-00020          13. delaunay_n24            14. rgg_n_2_24_s0           15. inf-road_usa"<< endl;
    cout << "16. nlpkkt240               17. inf-europe_osm" << endl;

    cout << endl;
    cout << "Enter Your Choice: ";
    cin >> fileNo;
    cout << endl;

    switch(fileNo){
        case 1:
            filename += "chesapeake";
            filenameforCorrection += "chesapeake";
            break;
        case 2:
            filename += "rgg_n_2_16_s0";
            filenameforCorrection += "rgg_n_2_16_s0";
            break;
        case 3:
            filename += "kron_g500-logn16";
            filenameforCorrection += "kron_g500-logn16";
            break;
        case 4:
            filename += "inf-luxembourg_osm";
            filenameforCorrection += "inf-luxembourg_osm";
            break;
        case 5:
            filename += "delaunay_n17";
            filenameforCorrection += "delaunay_n17";
            break;
        case 6:
            filename += "co-papers-citeseer";
            filenameforCorrection += "co-papers-citeseer";
            break;
        case 7:
            filename += "co-papers-dblp";
            filenameforCorrection += "co-papers-dblp";
            break;
        case 8:
            filename += "kron_g500-logn21";
            filenameforCorrection += "kron_g500-logn21";
            break;
        case 9:
            filename += "hugetrace-00000";
            filenameforCorrection += "hugetrace-00000";
            break;
        case 10:
            filename += "channel-500x100x100-b050";
            filenameforCorrection += "channel-500x100x100-b050";
            break;
        case 11:
            filename += "delaunay_n23";
            filenameforCorrection += "delaunay_n23";
            break;
        case 12:
            filename += "hugetrace-00020";
            filenameforCorrection += "hugetrace-00020";
            break;
        case 13:
            filename += "delaunay_n24";
            filenameforCorrection += "delaunay_n24";
            break;
        case 14:
            filename += "rgg_n_2_24_s0";
            filenameforCorrection += "rgg_n_2_24_s0";
            break;
        case 15:
            filename += "inf-road_usa";
            filenameforCorrection += "inf-road_usa";
            break;
        case 16:
            filename += "nlpkkt240";
            filenameforCorrection += "nlpkkt240";
            break;
        case 17:
            filename += "inf-europe_osm";
            filenameforCorrection += "inf-europe_osm";
            break;
        default:
            cout << "Invalid Choice." << endl;
            return 0;
    }

    if(fileNo == 3 || fileNo == 8 || fileNo == 16) weighted = 1;

    filename += ext;
    ifstream file(filename);

    if (!file.is_open())
    {
        cerr << "Failed to open the file." << endl;
        return 0;
    }

    cout << "What do you want to compute?" << endl;
    cout << "1. Vertex-Based SSSP" << endl;
    cout << "2. Edge-Based SSSP" << endl;
    cout << "3. Worklist-Based SSSP" << endl;
    cout << "4. Even Odd Thread Distributed Worklist Based SSSP" << endl;
    cout << "5. Balanced Worklist Based SSSP" << endl;
    cout << "6. Edge Centric Worklist Based SSSP" << endl;
    cout << "7. Bucket Based Worklist SSSP" << endl;
    cout << "8. Bucket Based Extended Worklist SSSP" << endl;   // Not Working
    cout << "9. Triangle Counting Vertex Centric" << endl;
    cout << "10. Triangle Counting Edge Centric" << endl;
    cout << "11. Triangle Counting Sorted Vertex Centric" << endl;
    cout << endl;

    cout << "Enter Your Choice: ";

    cin >> algoChoice;

    if(algoChoice >= 9 && algoChoice <= 11) filenameforCorrection = "../Gunrockresults/TC/" + filenameforCorrection + ".txt";
    else filenameforCorrection = "../Gunrockresults/SSSP/" + filenameforCorrection + ".txt";

    cout << endl;
    cout << "Graph: " << filename << endl;

    file.close();
//    filename = "input.txt";

    return 1;
}

void constructCSR(ll &vertices, ll *index, ll *headvertex, ll *weights, int directed, int weighted, vector<Edge> &edgeList, map<ll, ll> vertexCount, ll* vertexToIndexMap)
{
    ll edges = edgeList.size();

    // constructing indices for index array
    index[0] = 0;
    int i = 1;
    for(auto& p: vertexCount){
        index[i] = p.second;
        ++i;
    }

    i = 0;
    for(auto& p: vertexCount){
        vertexToIndexMap[i++] = p.first;
    }

    for (ll j = 1; j < vertices + 1; ++j)
        index[j] += index[j - 1];

    // constructing the headvertex and weights array
    for (ll j = 0; j < edges; ++j)
    {
        Edge e = edgeList[j];
        headvertex[j] = e.dest;
        weights[j] = e.wt;
    }
}

void printCSR(ll &vertices, ll *index, ll *headvertex, ll *weights, ll &edges, ll *vertexToIndexMap)
{
    cout << "----------------------------------STARTED PRINTING CSR---------------------------------" << endl;
//    cout << "Vertex Mapping: ";
//    for(int i = 0; i < vertices; ++i)
//        cout << vertexToIndexMap[i] << ' ';
//    cout << endl;

    cout << "Index: ";
    for(int i = 0; i < vertices + 1; ++i)
        cout << index[i] << ' ';
    cout << endl;

    cout << "Head Vertex: ";
    for (int i = 0; i < edges; ++i)
        cout << headvertex[i] << ' ';
    cout << endl;
//
//    cout << "Weights: ";
//    for (int i = 0; i < edges; ++i)
//        cout << weights[i] << ' ';
//    cout << endl;
}

void printEdgeList(vector<Edge> &edgeList)
{
    cout << "-------------------------------STARTED PRINTING EDGELIST------------------------------" << endl;
    for (Edge e : edgeList)
        cout << e.src << ' ' << e.dest << ' ' << e.wt << endl;
}

ll nearestPowerOf2(ll value) {
    if (value <= 0) {
        return 1;
    }

    ll exponent = round(log2(value));
    return pow(2, exponent);
}

void printTimings(vector<double>& timings){
    for(double i: timings) cout << i << endl;
}

void constructSrcCSR(ll &vertices, ll *index, ll *sources, ll *headvertex, ll *weights, int directed, int weighted, vector<Edge> &edgeList, map<ll, ll> vertexCount, ll* vertexToIndexMap)
{
    ll edges = edgeList.size();

    // constructing indices for index array
    index[0] = 0;
    int i = 1;
    for(auto& p: vertexCount){
        index[i] = p.second;
        ++i;
    }

    i = 0;
    for(auto& p: vertexCount){
        vertexToIndexMap[i++] = p.first;
    }

    for (ll j = 1; j < vertices + 1; ++j)
        index[j] += index[j - 1];

    // constructing the headvertex and weights array
    for (ll j = 0; j < edges; ++j)
    {
        Edge e = edgeList[j];
        sources[j] = e.src;
        headvertex[j] = e.dest;
        weights[j] = e.wt;
    }
}

void ssspBalancedWorklist(ll totalVertices, ll totalEdges, ll *dindex, ll *dheadvertex, ll *dweights, ll src, string &filenameforCorrection){
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    float totalTime = 0.0;
    float time;

    ll *dist;
    hipMalloc(&dist, (ll)(totalVertices) *sizeof(ll));

    cout << "Chosen source vertex is: " << src << endl;

    unsigned int nodeblocks = ceil((double)totalVertices / (double)BLOCKSIZE);

    time = 0.0;
    hipEventRecord(start);
    ssspVertexInit<<<nodeblocks, BLOCKSIZE>>>(totalVertices, dist);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&time, start, stop);
    totalTime += time;

    cout << "Initialized distance array" << endl;
    cout << endl;

    float *workers = (float*)malloc(sizeof(float));
    float *temp1 = (float*)malloc(sizeof(float));       //Host index for 1st worklist
    float *temp2 = (float*)malloc(sizeof(float));       //Host index for 2nd worklist

    *workers = 1;

    ll *curr;
    hipMalloc(&curr, (4 * totalVertices) * sizeof(ll));

    ll *next1;
    hipMalloc(&next1, (2 * totalVertices) * sizeof(ll));

    ll *next2;
    hipMalloc(&next2, (2 * totalVertices) * sizeof(ll));

    cout << "Initialized current worklist" << endl;
    cout << endl;

    float *idx1, *idx2;                 //Device indices for the worklists
    hipMalloc(&idx1, sizeof(float));
    hipMalloc(&idx2, sizeof(float));

    cout << "Defined indices for next worklists" << endl;
    cout << endl;

    time = 0.0;
    hipEventRecord(start);
    init<<<1,1>>>(src, dist, curr);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&time, start, stop);
    totalTime += time;

    cout << "Initialized source distance and current worklist" << endl;
    cout << endl;

    ll itr = 0;

    unsigned blocks = ceil((double)(*workers) / BLOCKSIZE);

    while(true){
        time = 0.0;
        hipEventRecord(start);
        setIndexForWorklist2<<<1, 1>>>(idx1, idx2);
        hipEventRecord(stop);
        hipEventSynchronize(stop);

        hipEventElapsedTime(&time, start, stop);
        totalTime += time;
        hipDeviceSynchronize();

        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            printf("CUDA Error2: %s\n", hipGetErrorString(err));
            return;
        }

        ll limit = *workers / 2;

        time = 0.0;
        hipEventRecord(start);
        ssspBalancedWorklistKernel<<<blocks, BLOCKSIZE>>>(2 * totalVertices, *workers, dindex, dheadvertex, dweights, curr, next1, next2, dist, idx1, idx2, limit);
        hipEventRecord(stop);
        hipEventSynchronize(stop);

        hipEventElapsedTime(&time, start, stop);
        totalTime += time;
        hipDeviceSynchronize();

        ++itr;

        err = hipGetLastError();
        if (err != hipSuccess) {
            printf("CUDA Error1: %s\n", hipGetErrorString(err));
            return;
        }

        time = 0.0;
        hipEventRecord(start);
        hipMemcpy(temp1, idx1, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(temp2, idx2, sizeof(int), hipMemcpyDeviceToHost);
        hipEventRecord(stop);
        hipEventSynchronize(stop);

        hipEventElapsedTime(&time, start, stop);
        totalTime += time;

        *workers = *temp1 + *temp2;

        if(*workers == 0) break;

        err = hipGetLastError();
        if (err != hipSuccess) {
            printf("CUDA Error3: %s\n", hipGetErrorString(err));
            return;
        }

        blocks = ceil((double) (*workers) / BLOCKSIZE);

        time = 0.0;
        hipEventRecord(start);
        mergeWorklist<<<blocks, BLOCKSIZE>>>(curr, next1, next2, idx1, idx2);
        hipEventRecord(stop);
        hipEventSynchronize(stop);

        hipEventElapsedTime(&time, start, stop);
        totalTime += time;

        err = hipGetLastError();
        if (err != hipSuccess) {
            printf("CUDA Error4: %s\n", hipGetErrorString(err));
            return;
        }

//        print2<<<1,1>>>(*workers, curr);
//        hipDeviceSynchronize();
        err = hipGetLastError();
        if (err != hipSuccess) {
            printf("CUDA Error5: %s\n", hipGetErrorString(err));
            return;
        }
    }

    cout << "Total Iterations: " << itr << endl;

    cout << "First 10 values of dist vector: ";
    printDist<<<1,1>>>(totalVertices, dist);
    hipDeviceSynchronize();

    cout << "Total Time: " << totalTime << endl;

    cout << endl;

    cout << "Checking correctness with vertex-centric approach..." << endl;

    ssspVertexCentricCorrectness(totalVertices, dindex, dheadvertex, dweights, src, dist);

    cout << "Checking correctness with SlabGraph results..." << endl;
    checkSSSPCorrectnessWithSlabGraph(dist, filenameforCorrection);
}

void ssspWorklist2(ll totalVertices, ll totalEdges, ll *dindex, ll *dheadvertex, ll *dweights, ll srcVertex, string &filenameforCorrection){
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    float totalTime = 0.0;
    float time;

    ll *dist;
    hipMalloc(&dist, (ll)(totalVertices) *sizeof(ll));

    cout << "Chosen source vertex is: " << srcVertex << endl;

    unsigned int nodeblocks = ceil((double)totalVertices / (double)BLOCKSIZE);

    time = 0.0;
    hipEventRecord(start);
    ssspVertexInit<<<nodeblocks, BLOCKSIZE>>>(totalVertices, dist);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&time, start, stop);
    totalTime += time;

    cout << "Initialized distance array" << endl;
    cout << endl;

    float *workers = (float*)malloc(sizeof(float));
    float *temp1 = (float*)malloc(sizeof(float));       //Host index for 1st worklist
    float *temp2 = (float*)malloc(sizeof(float));       //Host index for 2nd worklist

    *workers = 1;

    ll *curr;
    hipMalloc(&curr, 4 * totalVertices * sizeof(ll));

    ll *next1;
    hipMalloc(&next1, 2 * totalVertices * sizeof(ll));

    ll *next2;
    hipMalloc(&next2, 2 * totalVertices * sizeof(ll));

    cout << "Initialized current worklist" << endl;
    cout << endl;

    float *idx1, *idx2;
    hipMalloc(&idx1, sizeof(float));
    hipMalloc(&idx2, sizeof(float));

    cout << "Defined index for next worklist" << endl;
    cout << endl;

    time = 0.0;
    hipEventRecord(start);
    init<<<1,1>>>(srcVertex, dist, curr);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&time, start, stop);
    totalTime += time;

    cout << "Initialized source distance and current worklist" << endl;
    cout << endl;

    ll itr = 0;

    unsigned blocks = ceil((double)(*workers) / BLOCKSIZE);

    while(true){
        time = 0.0;
        hipEventRecord(start);
        setIndexForWorklist2<<<1, 1>>>(idx1, idx2);
        hipEventRecord(stop);
        hipEventSynchronize(stop);

        hipEventElapsedTime(&time, start, stop);
        totalTime += time;
        hipDeviceSynchronize();

        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            printf("CUDA Error2: %s\n", hipGetErrorString(err));
            return;
        }

        time = 0.0;
        hipEventRecord(start);
        ssspWorklistKernel2<<<blocks, BLOCKSIZE>>>(*workers, dindex, dheadvertex, dweights, curr, next1, next2, dist, idx1, idx2);
        hipEventRecord(stop);
        hipEventSynchronize(stop);

        hipEventElapsedTime(&time, start, stop);
        totalTime += time;
        hipDeviceSynchronize();

        ++itr;

        err = hipGetLastError();
        if (err != hipSuccess) {
            printf("CUDA Error1: %s\n", hipGetErrorString(err));
            return;
        }

        time = 0.0;
        hipEventRecord(start);
        hipMemcpy(temp1, idx1, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(temp2, idx2, sizeof(int), hipMemcpyDeviceToHost);
        hipEventRecord(stop);
        hipEventSynchronize(stop);

        hipEventElapsedTime(&time, start, stop);
        totalTime += time;

        *workers = *temp1 + *temp2;

        if(*workers == 0) break;

        err = hipGetLastError();
        if (err != hipSuccess) {
            printf("CUDA Error3: %s\n", hipGetErrorString(err));
            return;
        }

        blocks = ceil((double) (*workers) / BLOCKSIZE);

        time = 0.0;
        hipEventRecord(start);
        mergeWorklist<<<blocks, BLOCKSIZE>>>(curr, next1, next2, idx1, idx2);
        hipEventRecord(stop);
        hipEventSynchronize(stop);

        hipEventElapsedTime(&time, start, stop);
        totalTime += time;

        err = hipGetLastError();
        if (err != hipSuccess) {
            printf("CUDA Error4: %s\n", hipGetErrorString(err));
            return;
        }

//        print2<<<1,1>>>(*workers, curr);
//        hipDeviceSynchronize();
        err = hipGetLastError();
        if (err != hipSuccess) {
            printf("CUDA Error5: %s\n", hipGetErrorString(err));
            return;
        }
    }

    cout << "Total Iterations: " << itr << endl;

    cout << "First 10 values of dist vector: ";
    printDist<<<1,1>>>(totalVertices, dist);
    hipDeviceSynchronize();

    cout << "Total Time: " << totalTime << endl;

    cout << endl;

    cout << "Checking correctness with vertex-centric approach..." << endl;

    ssspVertexCentricCorrectness(totalVertices, dindex, dheadvertex, dweights, srcVertex, dist);

    cout << "Checking correctness with SlabGraph results..." << endl;
    checkSSSPCorrectnessWithSlabGraph(dist, filenameforCorrection);
}

void ssspWorklist(ll totalVertices, ll totalEdges, ll *dindex, ll *dheadvertex, ll *dweights, ll srcVertex, string &filenameforCorrection){
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    float totalTime = 0.0;
    float time;

    ll *dist;
    hipMalloc(&dist, (ll)(totalVertices) *sizeof(ll));

    cout << "Chosen source vertex is: " << srcVertex << endl;

    unsigned int nodeblocks = ceil((double)totalVertices / (double)BLOCKSIZE);

    ssspVertexInit<<<nodeblocks, BLOCKSIZE>>>(totalVertices, dist);
    hipDeviceSynchronize();
    cout << "Initialized distance array" << endl;
    cout << endl;

    float *workers = (float*)malloc(sizeof(float));
//    cout << "done";
    *workers = 1;

    ll *curr;
    hipMalloc(&curr, (3 * totalVertices) * sizeof(ll));

    ll *next;
    hipMalloc(&next, (3 * totalVertices) * sizeof(ll));

    cout << "Initialized current worklist" << endl;
    cout << endl;

    float *idx;
    hipMalloc(&idx, sizeof(float));

    cout << "Defined index for next worklist" << endl;
    cout << endl;

    time = 0.0;
    hipEventRecord(start);
    init<<<1,1>>>(srcVertex, dist, curr);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    totalTime += time;

    hipError_t err = hipGetLastError();
//    if (err != hipSuccess) {
//        printf("CUDA Error0: %s\n", hipGetErrorString(err));
//        return;
//    }

    cout << "Initialized source distance and current worklist" << endl;
    cout << endl;

    ll itr = 1;
    unsigned worklist_blocks;

    time = 0.0;
    hipEventRecord(start);
    while(true){
        worklist_blocks = ceil((double)(*workers) / BLOCKSIZE);

        setIndexForWorklist<<<1, 1>>>(idx);
        hipDeviceSynchronize();

//        err = hipGetLastError();
//        if (err != hipSuccess) {
//            printf("CUDA Error1: %s\n", hipGetErrorString(err));
//            return;
//        }

        if(itr % 2 != 0) {
            ssspWorklistKernel<<<worklist_blocks, BLOCKSIZE>>>(*workers, dindex, dheadvertex, dweights, curr, next, dist, idx, 3 * totalVertices);
            hipDeviceSynchronize();

//            err = hipGetLastError();
//            if (err != hipSuccess) {
//                printf("CUDA Error odd: %s\n", hipGetErrorString(err));
//                return;
//            }

//            print<<<1,1>>>(idx, next);
//            hipDeviceSynchronize();
        }
        else{
            ssspWorklistKernel<<<worklist_blocks, BLOCKSIZE>>>(*workers, dindex, dheadvertex, dweights, next, curr, dist, idx, 3 * totalVertices);
            hipDeviceSynchronize();

//            err = hipGetLastError();
//            if (err != hipSuccess) {
//                printf("CUDA Error even: %s\n", hipGetErrorString(err));
//                return;
//            }

//            print<<<1,1>>>(idx, curr);
//            hipDeviceSynchronize();
        }

        ++itr;

//        err = hipGetLastError();
//        if (err != hipSuccess) {
//            printf("CUDA Error2: %s\n", hipGetErrorString(err));
//            return;
//        }

        hipMemcpy(workers, idx, sizeof(int), hipMemcpyDeviceToHost);
        hipDeviceSynchronize();

        if(*workers == 0) break;

//        err = hipGetLastError();
//        if (err != hipSuccess) {
//            printf("CUDA Erro3: %s\n", hipGetErrorString(err));
//            return;
//        }
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    totalTime += time;
    hipDeviceSynchronize();

    cout << "Total Iterations: " << itr << endl;

    cout << "First 40 values of dist vector: ";
    printDist<<<1,1>>>(totalVertices, dist);
    hipDeviceSynchronize();

    cout << "Total Time: " << totalTime << endl;

    cout << endl;

    cout << "Checking correctness with vertex-centric approach..." << endl;

    ssspVertexCentricCorrectness(totalVertices, dindex, dheadvertex, dweights, srcVertex, dist);

    cout << "Checking correctness with SlabGraph results..." << endl;
    checkSSSPCorrectnessWithSlabGraph(dist, filenameforCorrection);
}

void ssspEdgeCentric(ll totalVertices, ll totalEdges, ll *src, ll *dest, ll *weights, ll srcVertex, string &filenameforCorrection){
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    float totalTime = 0.0;
    float time;

    ll *dist;
    hipMalloc(&dist, (ll)(totalVertices) *sizeof(ll));

    unsigned int nodeblocks = ceil((double)totalVertices / (double)BLOCKSIZE);

    time = 0.0;
    hipEventRecord(start);
    ssspVertexInit<<<nodeblocks, BLOCKSIZE>>>(totalVertices, dist);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&time, start, stop);
    totalTime += time;

    cout << "Initialized distance array" << endl;
    cout << endl;

    time = 0.0;
    hipEventRecord(start);
    initSrc<<<1,1>>>(srcVertex, dist);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&time, start, stop);
    totalTime += time;

    cout << "Initialized source distance" << endl;
    cout << endl;

    int *hchanged;
    hchanged = (int *)malloc(sizeof(int));

    int *dchanged;
    hipMalloc(&dchanged, sizeof(int));

    unsigned blocks = ceil((double)totalEdges / BLOCKSIZE);

    int itr = 1;

    while(true){
        *hchanged = 0;
        hipMemcpy(dchanged, hchanged, sizeof(int), hipMemcpyHostToDevice);

//        cout << "Launching Kernel: " << endl;

        time = 0.0;
        hipEventRecord(start);
        ssspEdgeCall<<<blocks, BLOCKSIZE>>>(totalEdges, src, dest, weights, dist, dchanged);
        hipEventRecord(stop);
        hipEventSynchronize(stop);

        hipEventElapsedTime(&time, start, stop);
        totalTime += time;

        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            printf("CUDA Error: %s\n", hipGetErrorString(err));
        }

        hipMemcpy(hchanged, dchanged, sizeof(int), hipMemcpyDeviceToHost);

//        cout << "Done Iteration: " << itr << endl;

        ++itr;

        if(*hchanged == 0) break;
    }

    cout << "Total Iterations: " << itr << endl;

    cout << "First 10 values of Device vector: ";
    printDist<<<1,1>>>(totalVertices, dist);
    hipDeviceSynchronize();

    cout << "Total Time: " << totalTime << endl;

    cout << "Checking correctness with SlabGraph results..." << endl;
    checkSSSPCorrectnessWithSlabGraph(dist, filenameforCorrection);
}

void printssspCpu(ll totalVertices, ll *dist){
    cout << "First 10 values of Host vector: ";
    for(ll i = 0; i < 10; ++i) cout << dist[i] << ' ';
    cout << endl;
}

void ssspSerial(ll totalVertices, ll *index, ll *headvertex, ll *weights, ll *dist, ll src){
    for(ll i = 0; i < totalVertices; ++i) dist[i] = INT_MAX;

    dist[src] = 0;

    int changed;
    while(true){
        changed = 0;
        for(ll u = 0; u < totalVertices; ++u){
            ll start = index[u];
            ll end = index[u + 1];

            for(ll i = start; i < end; ++i){
                ll v = headvertex[i];
                ll wt = weights[i];

                if(dist[v] > dist[u] + wt){
                    dist[v] = dist[u] + wt;
                    changed = 1;
                }
            }
        }

        if(changed == 0) break;
    }
}

void ssspVertexCentricCorrectness(ll totalVertices, ll *dindex, ll *dheadvertex, ll *dweights, ll srcVertex, ll *wdist){
    ll *vdist;      //dist vector for vertex centric approach. wdist is dist vector for worklist based approach
    hipMalloc(&vdist, (ll)(totalVertices) * sizeof(ll));

    unsigned int nodeblocks = ceil((double)totalVertices / (double)BLOCKSIZE);

    ssspVertexInit<<<nodeblocks, BLOCKSIZE>>>(totalVertices, vdist);
    hipDeviceSynchronize();

    initSrc<<<1,1>>>(srcVertex, vdist);
    hipDeviceSynchronize();

    int *hchanged;
    hchanged = (int *)malloc(sizeof(int));

    int *dchanged;
    hipMalloc(&dchanged, sizeof(int));

    while(true){
        *hchanged = 0;
        hipMemcpy(dchanged, hchanged, sizeof(int), hipMemcpyHostToDevice);

        ssspVertexCall<<<nodeblocks, BLOCKSIZE>>>(totalVertices, dindex, dheadvertex, dweights, vdist, dchanged);
        hipDeviceSynchronize();

        hipMemcpy(hchanged, dchanged, sizeof(int), hipMemcpyDeviceToHost);

        if(*hchanged == 0) break;
    }

    cout << "First 40 values of dist vector: ";
    printDist<<<1,1>>>(totalVertices, vdist);
    hipDeviceSynchronize();

    int *hequalityFlag;
    int *dequalityFlag;

    hequalityFlag = (int *)malloc(sizeof(int));
    hipMalloc(&dequalityFlag, sizeof(int));

    *hequalityFlag = 1;
    hipMemcpy(dequalityFlag, hequalityFlag, sizeof(int), hipMemcpyHostToDevice);

    checkCorrectness<<<nodeblocks, BLOCKSIZE>>>(totalVertices, vdist, wdist, dequalityFlag);
    hipDeviceSynchronize();

    hipMemcpy(hequalityFlag, dequalityFlag, sizeof(int), hipMemcpyDeviceToHost);
    if(*hequalityFlag == 1) cout << "Correctness Verified!" << endl;
    else cout << "Incorrect Result!" << endl;
}

void ssspVertexCentric(ll totalVertices, ll *dindex, ll *dheadvertex, ll *dweights, ll src, string &filenameforCorrection){
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    float totalTime = 0.0;
    float time;

    ll *dist;
    hipMalloc(&dist, (ll)(totalVertices) *sizeof(ll));

    cout << "Chosen source vertex is: " << src << endl;

    unsigned int nodeblocks = ceil((double)totalVertices / (double)BLOCKSIZE);

    time = 0.0;
    hipEventRecord(start);
    ssspVertexInit<<<nodeblocks, BLOCKSIZE>>>(totalVertices, dist);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&time, start, stop);
    totalTime += time;

    cout << "Initialized distance array" << endl;
    cout << endl;

    time = 0.0;
    hipEventRecord(start);
    initSrc<<<1,1>>>(src, dist);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&time, start, stop);
    totalTime += time;

    cout << "Initialized source distance" << endl;
    cout << endl;

    int *hchanged;
    hchanged = (int *)malloc(sizeof(int));

    int *dchanged;
    hipMalloc(&dchanged, sizeof(int));

    int itr = 1;

    while(true){
        *hchanged = 0;
        hipMemcpy(dchanged, hchanged, sizeof(int), hipMemcpyHostToDevice);

//        cout << "Launching Kernel: " << endl;

        time = 0.0;
        hipEventRecord(start);
        ssspVertexCall<<<nodeblocks, BLOCKSIZE>>>(totalVertices, dindex, dheadvertex, dweights, dist, dchanged);
        hipEventRecord(stop);
        hipEventSynchronize(stop);

        hipEventElapsedTime(&time, start, stop);
        totalTime += time;

        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            printf("CUDA Error: %s\n", hipGetErrorString(err));
        }

        hipMemcpy(hchanged, dchanged, sizeof(int), hipMemcpyDeviceToHost);

//        cout << "Done Iteration: " << itr << endl;

        ++itr;

        if(*hchanged == 0) break;
    }

    cout << "Total Iterations: " << itr << endl;

    cout << "First 10 values of dist vector: ";
    printDist<<<1,1>>>(totalVertices, dist);
    hipDeviceSynchronize();

    cout << "Total Time: " << totalTime << endl;

    cout << "Checking correctness with SlabGraph results..." << endl;
    checkSSSPCorrectnessWithSlabGraph(dist, filenameforCorrection);
}

void ssspEdgeWorklistCentric(ll totalvertices, ll totalEdges, ll *csr_offsets, ll *csr_edges, ll *csr_weights, ll srcVertex, string &filenameforCorrection){
    // Timing Calculations
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    float totalTime = 0.0;
    float time;

    // Allocating space on device for distance vector to store the distances
    ll *dist;
    hipMalloc(&dist, (ll)(totalvertices) * sizeof(ll));
    cout << "Space allocated for distance vector on device." << endl;
    cout << endl;

    // Initialising distance vector
    unsigned int nodeblocks = ceil((double)totalvertices / (double)BLOCKSIZE);

//    time = 0.0;
//    hipEventRecord(start);
    ssspVertexInit<<<nodeblocks, BLOCKSIZE>>>(totalvertices, dist);
    hipDeviceSynchronize();
//    hipEventRecord(stop);
//    hipEventSynchronize(stop);
//    hipEventElapsedTime(&time, start, stop);
//    totalTime += time;

    cout << "Initialized distance array. Chosen source vertex is: " << srcVertex << endl;
    cout << endl;

    // Allocating space on device for input frontier. Size = number of vertices
    ll *input_frontier;
    hipMalloc(&input_frontier, (ll)(2 * totalvertices) * sizeof(ll));
    cout << "Space allocated for input frontiers." << endl;
    cout << endl;

    ll *deg_for_input_frontier;
    hipMalloc(&deg_for_input_frontier, (ll)(2 * totalvertices) * sizeof(ll));

    ll *frontier_offset;
    hipMalloc(&frontier_offset, (ll)(2 * totalvertices + 1) * sizeof(ll));
    cout << "Space allocated for frontier offset." << endl;
    cout << endl;

    // Allocating space on device for output frontier. Size = number of vertices + 1
    ll *output_frontier;
    hipMalloc(&output_frontier, (ll)(2 * totalvertices) * sizeof(ll));
    cout << "Space allocated for output frontier." << endl;
    cout << endl;

    // Defining global index that can operate on input frontier.
    float *idx;
    hipMalloc(&idx, sizeof(float));
    cout << "Defined index for input frontier." << endl;
    cout << endl;

    // Initialising distance of source vertex and adding source vertex to input frontier.
    time = 0.0;
    hipEventRecord(start);
    init<<<1,1>>>(srcVertex, dist, input_frontier);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    totalTime += time;

    hipError_t err = hipGetLastError();                   // Catching errors, if any.
    if (err != hipSuccess) {
        printf("CUDA Error while initialising distance for source vertex: %s\n", hipGetErrorString(err));
        return;
    }
    cout << "Initialized source distance and added source vertex to input frontier." << endl;
    cout << endl;

    // Defining number of workers. Initializing it with 1.
    float *workers = (float*)malloc(sizeof(float));
    *workers = 1;

    ll iterations = 1;  // For calculating total iterations

    // Meta data for computing frontier offset.
    ll *host_prefix_sum;
    host_prefix_sum = (ll *)malloc(sizeof(ll));
    ll *device_prefix_sum;
    hipMalloc(&device_prefix_sum, sizeof(ll));

    // Declaring device ptrs for device arrays.
    thrust::device_ptr<ll> thrust_input_ptr;
    thrust::device_ptr<ll> thrust_output_ptr;

    ll frontier_size;
    unsigned sssp_kernel_blocks;
    unsigned degree_blocks;
    float prefixSumTime = 0.0;
    clock_t time_req;

    // Normal SSSP loop
    time = 0.0;
    hipEventRecord(start);
    while(true){
        // Setting index of the frontier.
        setIndexForWorklist<<<1, 1>>>(idx);
        hipDeviceSynchronize();

//        err = hipGetLastError();
//        if (err != hipSuccess) {
//            printf("CUDA Error while setting index for frontier: %s\n", hipGetErrorString(err));
//            return;
//        }

        if(iterations % 2 != 0){
            /** Constructing Frontier offset **/
//            cout << "iteration: " << iterations << endl;

            // Allocating frontier_size to number of current workers
            frontier_size = *workers;
//            cout << "Size: " << frontier_size << endl;

            // Replacing nodes present in input_frontier with their respective degrees
            degree_blocks = ceil((double) (frontier_size) / BLOCKSIZE);
            time_req = clock();
            replaceNodeWithDegree<<<degree_blocks, BLOCKSIZE>>>(csr_offsets, input_frontier, deg_for_input_frontier, frontier_size);
            hipDeviceSynchronize();

//            err = hipGetLastError();
//            if(err != hipSuccess){
//                printf("CUDA Error while replacing nodes with their degrees in frontier in odd iteration: %s\n", hipGetErrorString(err));
//                return;
//            }

            // Assigning device pointers to device arrays
            thrust_input_ptr = thrust::device_pointer_cast(deg_for_input_frontier);
            thrust_output_ptr = thrust::device_pointer_cast(frontier_offset);

            thrust::exclusive_scan(thrust::device, thrust_input_ptr, thrust_input_ptr + frontier_size + 1, thrust_output_ptr);
            hipDeviceSynchronize();
            time_req = clock() - time_req;

//            constructFrontierOffset<<<1,1>>>(csr_offsets, input_frontier, frontier_offset, frontier_size, device_prefix_sum);

//            err = hipGetLastError();
//            if(err != hipSuccess){
//                printf("CUDA Error while constructing frontier offset in odd iteration: %s\n", hipGetErrorString(err));
//                return;
//            }

            /** Copying device prefix sum to host **/
//            hipMemcpy(host_prefix_sum, device_prefix_sum, sizeof(ll), hipMemcpyDeviceToHost);
//            hipDeviceSynchronize();
            device_prefix_sum = frontier_offset + frontier_size;
            hipMemcpy(host_prefix_sum, device_prefix_sum, sizeof(ll), hipMemcpyDeviceToHost);
            hipDeviceSynchronize();

//            cout << "Prefix Sum: " << *host_prefix_sum << endl;

            // Threads to be launched will be equal to prefix sum
            sssp_kernel_blocks = ceil((double) (*host_prefix_sum) / BLOCKSIZE);

            /** Launching the SSSP edge centric kernel **/
            ssspEdgeWorklist<<<sssp_kernel_blocks, BLOCKSIZE>>>(csr_offsets, csr_edges, csr_weights, input_frontier, frontier_offset, output_frontier, device_prefix_sum, dist, idx, frontier_size);
            hipDeviceSynchronize();

//            err = hipGetLastError();
//            if(err != hipSuccess){
//                printf("CUDA Error while computing distance error in sssp kernel in odd iteration: %s\n", hipGetErrorString(err));
//                return;
//            }
        }
        else{
            /** Constructing Frontier offset **/

            // Allocating frontier_size to number of current workers
            frontier_size = *workers;

            // Replacing nodes present in input_frontier with their respective degrees
            degree_blocks = ceil((double) (frontier_size) / BLOCKSIZE);
            time_req = clock();
            replaceNodeWithDegree<<<degree_blocks, BLOCKSIZE>>>(csr_offsets, output_frontier, deg_for_input_frontier, frontier_size);
            hipDeviceSynchronize();

//            err = hipGetLastError();
//            if(err != hipSuccess){
//                printf("CUDA Error while replacing nodes with their degrees in frontier in even iteration: %s\n", hipGetErrorString(err));
//                return;
//            }

            // Assigning device pointers to device arrays
            thrust_input_ptr = thrust::device_pointer_cast(deg_for_input_frontier);
            thrust_output_ptr = thrust::device_pointer_cast(frontier_offset);

//          constructFrontierOffset<<<1,1>>>(csr_offsets, output_frontier, frontier_offset, *workers, device_prefix_sum);
            thrust::exclusive_scan(thrust::device, thrust_input_ptr, thrust_input_ptr + frontier_size + 1, thrust_output_ptr);
            hipDeviceSynchronize();
            time_req = clock() - time_req;

//            err = hipGetLastError();
//            if(err != hipSuccess){
//                printf("CUDA Error while constructing frontier offset in even iteration: %s\n", hipGetErrorString(err));
//                return;
//            }

            /** Copying device prefix sum to host **/
//            hipMemcpy(host_prefix_sum, device_prefix_sum, sizeof(ll), hipMemcpyDeviceToHost);
//            hipDeviceSynchronize();
            device_prefix_sum = frontier_offset + frontier_size;
            hipMemcpy(host_prefix_sum, device_prefix_sum, sizeof(ll), hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
//            cout << "Prefix Sum: " << *host_prefix_sum << endl;

            // Threads to be launched will be equal to prefix sum
            sssp_kernel_blocks = ceil((double) (*host_prefix_sum) / BLOCKSIZE);

            /** Launching the SSSP edge centric kernel **/
            ssspEdgeWorklist<<<sssp_kernel_blocks, BLOCKSIZE>>>(csr_offsets, csr_edges, csr_weights, output_frontier, frontier_offset, input_frontier, device_prefix_sum, dist, idx, frontier_size);
            hipDeviceSynchronize();

//            err = hipGetLastError();
//            if(err != hipSuccess){
//                printf("CUDA Error while computing distance error in sssp kernel in even iteration: %s\n", hipGetErrorString(err));
//                return;
//            }
        }

        ++iterations;
        prefixSumTime += ((float)time_req / CLOCKS_PER_SEC * 1000);

        // Copying the number of next workers to host.
        hipMemcpy(workers, idx, sizeof(float), hipMemcpyDeviceToHost);
        hipDeviceSynchronize();

        if(*workers == 0) break;

//        err = hipGetLastError();
//        if(err != hipSuccess){
//            printf("CUDA Error while copying device index to host workers: %s\n", hipGetErrorString(err));
//            return;
//        }
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    totalTime += time;
    hipDeviceSynchronize();


    cout << "Total Iterations: " << iterations << endl;
    cout << "Source Vertex: " << srcVertex << endl;
    cout << "First 40 values of dist vector: ";
    printDist<<<1,1>>>(totalvertices, dist);
    hipDeviceSynchronize();
    cout << "Total Time: " << totalTime << endl;
    cout << "Time for Prefix Sum Calculation: " << prefixSumTime << endl;
    cout << endl;

    cout << "Checking correctness with vertex-centric approach..." << endl;
    ssspVertexCentricCorrectness(totalvertices, csr_offsets, csr_edges, csr_weights, srcVertex, dist);

    cout << "Checking correctness with SlabGraph results..." << endl;
    checkSSSPCorrectnessWithSlabGraph(dist, filenameforCorrection);
}

void ssspBucketWorklist(ll totalvertices, ll totaledges, ll *csr_offsets, ll *csr_edges, ll *csr_weights, ll srcVertex, string &filenameforCorrection){
    // Timing Calculations
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    float totalTime = 0.0;
    float time;

    // Allocating space on device for distance vector to store the distances
    ll *dist;
    hipMalloc(&dist, (ll)(totalvertices) * sizeof(ll));
    cout << "Space allocated for distance vector on device." << endl;
    cout << endl;

    // Initialising distance vector
    unsigned int nodeblocks = ceil((double)totalvertices / (double)BLOCKSIZE);
    ssspVertexInit<<<nodeblocks, BLOCKSIZE>>>(totalvertices, dist);
    hipDeviceSynchronize();

    cout << "Initialized distance array. Chosen source vertex is: " << srcVertex << endl;
    cout << endl;

    // Allocating space on device for worklists. Size = number of vertices
    ll *curr1, *curr2, *next1, *next2;
    hipMalloc(&curr1, (ll)(2 * totalvertices) * sizeof(ll));
    hipMalloc(&curr2, (ll)(2 * totalvertices) * sizeof(ll));
    hipMalloc(&next1, (ll)(2 * totalvertices) * sizeof(ll));
    hipMalloc(&next2, (ll)(2 * totalvertices) * sizeof(ll));
    cout << "Space allocated for worklists." << endl;
    cout << endl;

    // Defining global index that can operate on worklists.
    float *idx1, *idx2;
    hipMalloc(&idx1, sizeof(float));
    hipMalloc(&idx2, sizeof(float));
    cout << "Defined index for input frontier." << endl;
    cout << endl;

    // Initialising distance of source vertex and adding source vertex to input frontier.
    time = 0.0;
    hipEventRecord(start);
    init<<<1,1>>>(srcVertex, dist, curr1);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    totalTime += time;

    hipError_t err = hipGetLastError();                   // Catching errors, if any.
    if (err != hipSuccess) {
        printf("CUDA Error while initialising distance for source vertex: %s\n", hipGetErrorString(err));
        return;
    }
    cout << "Initialized source distance and added source vertex to input frontier." << endl;
    cout << endl;

    // Defining number of workers. Initializing it with 1.
    float *workers1 = (float*)malloc(sizeof(float));
    float *workers2 = (float*)malloc(sizeof(float));
    *workers1 = 1;
    *workers2 = 0;

    ll iterations = 1;  // For calculating total iterations

    unsigned worklist1_blocks, worklist2_blocks;

    // Normal SSSP loop
    time = 0.0;
    hipEventRecord(start);
    while(true){
        // Setting index of the frontier.
        setIndexForWorklist2<<<1, 1>>>(idx1, idx2);
        hipDeviceSynchronize();

        err = hipGetLastError();
        if (err != hipSuccess) {
            printf("CUDA Error while setting indices for worklists: %s\n", hipGetErrorString(err));
            return;
        }

        // Calculating the total blocks required for both kernels
        worklist1_blocks = ceil((double) (*workers1) / BLOCKSIZE);
        worklist2_blocks = ceil((double) (*workers2) / BLOCKSIZE);

        if(iterations % 2 != 0){
            /** Checking whether blocks are positive else no need to launch the kernel. **/
            // ODD iteration: Reading from curr_j worklist and writing to next1 or next2 worklist
            if(worklist1_blocks > 0) ssspBucketWorklistKernel<<<worklist1_blocks, BLOCKSIZE>>>(*workers1, csr_offsets, csr_edges, csr_weights, curr1, next1, next2, dist, idx1, idx2);
            if(worklist2_blocks > 0) ssspBucketWorklistKernel<<<worklist2_blocks, BLOCKSIZE>>>(*workers2, csr_offsets, csr_edges, csr_weights, curr2, next1, next2, dist, idx1, idx2);
        }
        else{
            // EVEN iteration: Reading from next_j worklist and writing to curr1 or curr2 worklist
            if(worklist1_blocks > 0) ssspBucketWorklistKernel<<<worklist1_blocks, BLOCKSIZE>>>(*workers1, csr_offsets, csr_edges, csr_weights, next1, curr1, curr2, dist, idx1, idx2);
            if(worklist2_blocks > 0) ssspBucketWorklistKernel<<<worklist2_blocks, BLOCKSIZE>>>(*workers2, csr_offsets, csr_edges, csr_weights, next2, curr1, curr2, dist, idx1, idx2);
        }
        hipDeviceSynchronize();

        err = hipGetLastError();
        if (err != hipSuccess) {
            printf("CUDA Error for sssp Kernel: %s\n", hipGetErrorString(err));
            return;
        }

        // 1 Iteration completed!
        ++iterations;

        // Copying the number of next workers to host.
        hipMemcpy(workers1, idx1, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(workers2, idx2, sizeof(int), hipMemcpyDeviceToHost);
        hipDeviceSynchronize();

        err = hipGetLastError();
        if (err != hipSuccess) {
            printf("CUDA Error while copying device indices to host workers: %s\n", hipGetErrorString(err));
            return;
        }

        if(*workers1 == 0 && *workers2 == 0) break;
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    totalTime += time;
    hipDeviceSynchronize();

    cout << "Total Iterations: " << iterations << endl;
    cout << "Source Vertex: " << srcVertex << endl;
    cout << "First 40 values of dist vector: ";
    printDist<<<1,1>>>(totalvertices, dist);
    hipDeviceSynchronize();
    cout << "Total Time: " << totalTime << endl;
    cout << endl;

    cout << "Checking correctness with vertex-centric approach..." << endl;
    ssspVertexCentricCorrectness(totalvertices, csr_offsets, csr_edges, csr_weights, srcVertex, dist);

    cout << "Checking correctness with SlabGraph results..." << endl;
    checkSSSPCorrectnessWithSlabGraph(dist, filenameforCorrection);
}

void ssspBucketWorklist2(ll totalvertices, ll totaledges, ll *csr_offsets, ll *csr_edges, ll *csr_weights, ll srcVertex, string &filenameforCorrection){
    // Timing Calculations
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    float totalTime = 0.0;
    float time;

    // Allocating space on device for distance vector to store the distances
    ll *dist;
    hipMalloc(&dist, (ll)(totalvertices) * sizeof(ll));
    cout << "Space allocated for distance vector on device." << endl;
    cout << endl;

    // Initialising distance vector
    unsigned int nodeblocks = ceil((double)totalvertices / (double)BLOCKSIZE);
    ssspVertexInit<<<nodeblocks, BLOCKSIZE>>>(totalvertices, dist);
    hipDeviceSynchronize();

    cout << "Initialized distance array. Chosen source vertex is: " << srcVertex << endl;
    cout << endl;

    // Allocating space on device for worklists. Size = number of vertices
    ll *curr1, *curr2, *curr3, *next1, *next2, *next3;
    hipMalloc(&curr1, (ll)(2 * totalvertices) * sizeof(ll));
    hipMalloc(&curr2, (ll)(2 * totalvertices) * sizeof(ll));
    hipMalloc(&curr3, (ll)(2 * totalvertices) * sizeof(ll));
    hipMalloc(&next1, (ll)(2 * totalvertices) * sizeof(ll));
    hipMalloc(&next2, (ll)(2 * totalvertices) * sizeof(ll));
    hipMalloc(&next3, (ll)(2 * totalvertices) * sizeof(ll));
    cout << "Space allocated for worklists." << endl;
    cout << endl;

    // Defining global index that can operate on worklists.
    float *idx1, *idx2, *idx3;
    hipMalloc(&idx1, sizeof(float));
    hipMalloc(&idx2, sizeof(float));
    hipMalloc(&idx3, sizeof(float));
    cout << "Defined index for input frontier." << endl;
    cout << endl;

    // Initialising distance of source vertex and adding source vertex to input frontier.
    time = 0.0;
    hipEventRecord(start);
    init<<<1,1>>>(srcVertex, dist, curr1);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    totalTime += time;

    hipError_t err = hipGetLastError();                   // Catching errors, if any.
    if (err != hipSuccess) {
        printf("CUDA Error while initialising distance for source vertex: %s\n", hipGetErrorString(err));
        return;
    }
    cout << "Initialized source distance and added source vertex to input frontier." << endl;
    cout << endl;

    // Defining number of workers. Initializing it with 1.
    float *workers1 = (float*)malloc(sizeof(float));
    float *workers2 = (float*)malloc(sizeof(float));
    float *workers3 = (float*)malloc(sizeof(float));
    *workers1 = 1;
    *workers2 = 0;
    *workers3 = 0;

    ll iterations = 1;  // For calculating total iterations

    unsigned worklist1_blocks, worklist2_blocks, worklist3_blocks;

    // Normal SSSP loop
    time = 0.0;
    hipEventRecord(start);
    while(true){
        // Setting index of the frontier.
        setIndexForWorklist<<<1, 1>>>(idx1, idx2, idx3);
        hipDeviceSynchronize();

        err = hipGetLastError();
        if (err != hipSuccess) {
            printf("CUDA Error while setting indices for worklists: %s\n", hipGetErrorString(err));
            return;
        }

        // Calculating the total blocks required for both kernels
        worklist1_blocks = ceil((double) (*workers1) / BLOCKSIZE);
        worklist2_blocks = ceil((double) (*workers2) / BLOCKSIZE);
        worklist3_blocks = ceil((double) (*workers3) / BLOCKSIZE);

        if(iterations % 2 != 0){
            /** Checking whether blocks are positive else no need to launch the kernel. **/
            // ODD iteration: Reading from curr_j worklist and writing to next1 or next2 worklist
            if(worklist1_blocks > 0) ssspBucketWorklistKernel2<<<worklist1_blocks, BLOCKSIZE>>>(*workers1, csr_offsets, csr_edges, csr_weights, curr1, next1, next2, next3, dist, idx1, idx2, idx3);
            hipDeviceSynchronize();
            if(worklist2_blocks > 0) ssspBucketWorklistKernel2<<<worklist2_blocks, BLOCKSIZE>>>(*workers2, csr_offsets, csr_edges, csr_weights, curr2, next1, next2, next3, dist, idx1, idx2, idx3);
            hipDeviceSynchronize();
            if(worklist3_blocks > 0) ssspBucketWorklistKernel2<<<worklist3_blocks, BLOCKSIZE>>>(*workers3, csr_offsets, csr_edges, csr_weights, curr3, next1, next2, next3, dist, idx1, idx2, idx3);
            hipDeviceSynchronize();

        }
        else{
            // EVEN iteration: Reading from next_j worklist and writing to curr1 or curr2 worklist
            if(worklist1_blocks > 0) ssspBucketWorklistKernel2<<<worklist1_blocks, BLOCKSIZE>>>(*workers1, csr_offsets, csr_edges, csr_weights, next1, curr1, curr2, curr3, dist, idx1, idx2, idx3);
            hipDeviceSynchronize();
            if(worklist2_blocks > 0) ssspBucketWorklistKernel2<<<worklist2_blocks, BLOCKSIZE>>>(*workers2, csr_offsets, csr_edges, csr_weights, next2, curr1, curr2, curr3, dist, idx1, idx2, idx3);
            hipDeviceSynchronize();
            if(worklist3_blocks > 0) ssspBucketWorklistKernel2<<<worklist3_blocks, BLOCKSIZE>>>(*workers3, csr_offsets, csr_edges, csr_weights, next3, curr1, curr2, curr3, dist, idx1, idx2, idx3);
            hipDeviceSynchronize();
        }
        hipDeviceSynchronize();

        err = hipGetLastError();
        if (err != hipSuccess) {
            printf("CUDA Error for sssp Kernel: %s\n", hipGetErrorString(err));
            return;
        }

        // 1 Iteration completed!
        ++iterations;

        // Copying the number of next workers to host.
        hipMemcpy(workers1, idx1, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(workers2, idx2, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(workers3, idx2, sizeof(int), hipMemcpyDeviceToHost);
        hipDeviceSynchronize();

        err = hipGetLastError();
        if (err != hipSuccess) {
            printf("CUDA Error while copying device indices to host workers: %s\n", hipGetErrorString(err));
            return;
        }

        if(*workers1 == 0 && *workers2 == 0 && *workers3 == 0) break;
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    totalTime += time;
    hipDeviceSynchronize();

    cout << "Total Iterations: " << iterations << endl;
    cout << "Source Vertex: " << srcVertex << endl;
    cout << "First 40 values of dist vector: ";
    printDist<<<1,1>>>(totalvertices, dist);
    hipDeviceSynchronize();
    cout << "Total Time: " << totalTime << endl;
    cout << endl;

    cout << "Checking correctness with vertex-centric approach..." << endl;
    ssspVertexCentricCorrectness(totalvertices, csr_offsets, csr_edges, csr_weights, srcVertex, dist);

    cout << "Checking correctness with SlabGraph results..." << endl;
    checkSSSPCorrectnessWithSlabGraph(dist, filenameforCorrection);
}

void buildCOO(ll edges, vector<Edge>& edgelist, ll *src, ll *dest, ll *weights){
    for(ll i = 0; i < edges; ++i){
        Edge& e = edgelist[i];
        ll u = e.src;
        ll v = e.dest;
        ll wt = e.wt;

        src[i] = u;
        dest[i] = v;
        weights[i] = wt;
    }
}

void buildCSR(ll vertices, ll edges, vector<Edge>& edgelist, ll *index, ll *headvertex, ll *weights, unordered_map<ll, ll>& degrees){
    index[0] = 0;

    for(ll i = 0; i < edges; ++i){
        Edge& e = edgelist[i];
        ll u = e.src;
        ll v = e.dest;
        ll wt = e.wt;

        index[u + 1] = degrees[u];
        headvertex[i] = v;
        weights[i] = wt;
    }

    for(ll u = 1; u < vertices + 1; ++u) index[u] += index[u - 1];
}

void checkTCCorrectnessWithSlabGraph(unsigned int *wdist, string &filename){
    // Source Vertex should be 0
    if(filename == "../Gunrockresults/SSSP/nlpkkt240.txt") {
        cout << "Results are not available." << endl;
        return;
    }

    unsigned int *vdist;
    hipMalloc(&vdist, (40) * sizeof (unsigned int));

    ifstream file(filename); // replace with your file name
    string line;
    vector<ll> numbers;

    if (file.is_open()) {
        if (getline(file, line)) {
            istringstream iss(line);
            int num;
            while (iss >> num) {
                numbers.push_back(num);
            }
        }
        file.close();
    }

    unsigned int vectorsize = numbers.size();

    unsigned int *temp;
    temp = (unsigned int *) malloc(vectorsize * sizeof(unsigned int));

    for(int i = 0; i < vectorsize; ++i){
        temp[i] = numbers[i];
    }

    cout << endl;

    hipMemcpy(vdist, temp, (vectorsize) * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    int *hequalityFlag;
    int *dequalityFlag;

    hequalityFlag = (int *)malloc(sizeof(int));
    hipMalloc(&dequalityFlag, sizeof(int));

    *hequalityFlag = 1;
    hipMemcpy(dequalityFlag, hequalityFlag, sizeof(int), hipMemcpyHostToDevice);

    unsigned int nodeblocks = ceil((double)vectorsize / (double)BLOCKSIZE);

    checkCorrectness<<<nodeblocks, BLOCKSIZE>>>(vectorsize, vdist, wdist, dequalityFlag);
    hipDeviceSynchronize();

    cout << "First 40 values of TC: ";
    printTC<<<1,1>>>(vectorsize, vdist);
    hipDeviceSynchronize();

    cout << endl;

    hipMemcpy(hequalityFlag, dequalityFlag, sizeof(int), hipMemcpyDeviceToHost);
    if(*hequalityFlag == 1) cout << "Correctness Verified with SlabGraph!" << endl;
    else cout << "Incorrect Result!" << endl;
}

void checkSSSPCorrectnessWithSlabGraph(ll *wdist, string &filename){
    // Source Vertex should be 0
    if(filename == "../Gunrockresults/SSSP/nlpkkt240.txt") {
        cout << "Results are not available." << endl;
        return;
    }

    ll *vdist;
    hipMalloc(&vdist, (40) * sizeof (ll));

    ifstream file(filename); // replace with your file name
    string line;
    vector<ll> numbers;

    if (file.is_open()) {
        if (getline(file, line)) {
            istringstream iss(line);
            int num;
            while (iss >> num) {
                numbers.push_back(num);
            }
        }
        file.close();
    }

    ll vectorsize = numbers.size();

    ll *temp;
    temp = (ll *) malloc(vectorsize * sizeof(ll));

    for(int i = 0; i < vectorsize; ++i){
        temp[i] = numbers[i];
    }

    hipMemcpy(vdist, temp, (vectorsize) * sizeof(ll), hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    int *hequalityFlag;
    int *dequalityFlag;

    hequalityFlag = (int *)malloc(sizeof(int));
    hipMalloc(&dequalityFlag, sizeof(int));

    *hequalityFlag = 1;
    hipMemcpy(dequalityFlag, hequalityFlag, sizeof(int), hipMemcpyHostToDevice);

    unsigned int nodeblocks = ceil((double)vectorsize / (double)BLOCKSIZE);

    checkCorrectness<<<nodeblocks, BLOCKSIZE>>>(vectorsize, vdist, wdist, dequalityFlag);
    hipDeviceSynchronize();

    cout << "First 40 values of TC: ";
    printDist<<<1,1>>>(vectorsize, vdist);
    hipDeviceSynchronize();

    cout << endl;

    hipMemcpy(hequalityFlag, dequalityFlag, sizeof(int), hipMemcpyDeviceToHost);
    if(*hequalityFlag == 1) cout << "Correctness Verified with SlabGraph!" << endl;
    else cout << "Incorrect Result!" << endl;
}

void triangleCount(ll totalvertices, ll totaledges, ll *csr_offsets, ll *csr_edges, string &filenameforCorrection){
    // Timing Calculations
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    float totalTime = 0.0;
    float time;

    // Host side Triangle Count
    float *host_tc = (float *)malloc(sizeof(float *));

    // Device side Triangle Count
    float *device_tc;
    hipMalloc(&device_tc, sizeof(float));

    ll *device_tc_array;
    hipMalloc(&device_tc_array, (totalvertices) * sizeof(ll));

    unsigned blocks = ceil((double) totalvertices / BLOCKSIZE);

    cout << endl;
    cout << "Launching TC Kernel" << endl;

    // Kernel for TC
    time = 0.0;
    hipEventRecord(start);
    triangleCountVertexCentric<<<blocks, BLOCKSIZE>>>(csr_offsets, csr_edges, totalvertices, device_tc_array);
    hipDeviceSynchronize();

//    divideTCbysix<<<1,1>>>(device_tc);
//    hipDeviceSynchronize();

//    hipMemcpy(host_tc, device_tc, sizeof(float), hipMemcpyDeviceToHost);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    totalTime += time;
    hipDeviceSynchronize();

    cout << "Finished Kernel" << endl;

//    cout << "Triangle Count: " << *host_tc << endl;
    cout << "Total Time: " << totalTime << endl;
    cout << "First 40 values of TC: ";
    printDist<<<1,1>>>(totalvertices, device_tc_array);
    hipDeviceSynchronize();
    cout << endl;

    cout << "Checking Correctness with Gunrock..." << endl;
    checkSSSPCorrectnessWithSlabGraph(device_tc_array, filenameforCorrection);
}

void triangleCountEdgeCentric(ll totalvertices, ll totaledges, ll *csr_offsets, ll *csr_edges,  string &filenameforCorrection){
    // Timing Calculations
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    float totalTime = 0.0;
    float time;

    unsigned int *device_tc_array;
    hipMalloc(&device_tc_array, (totalvertices) * sizeof(unsigned int));

    unsigned blocks = ceil((double) totaledges / BLOCKSIZE);

    cout << endl;
    cout << "Launching Edge Centric TC Kernel" << endl;

    // Kernel for TC
    time = 0.0;
    hipEventRecord(start);
    triangleCountEdgeCentricKernel<<<blocks, BLOCKSIZE>>>(csr_offsets, csr_edges, device_tc_array, totaledges, totalvertices);
    hipDeviceSynchronize();

    blocks = ceil((double) totalvertices / BLOCKSIZE);
    divideTCArray<<<blocks, BLOCKSIZE>>>(device_tc_array, 2, totalvertices);
    hipDeviceSynchronize();

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    totalTime += time;
    hipDeviceSynchronize();

    cout << "Finished Kernel" << endl;

    cout << "Total Time: " << totalTime << endl;
    cout << "First 40 values of TC: ";
    printTC<<<1,1>>>(totalvertices, device_tc_array);
    hipDeviceSynchronize();
    cout << endl;

    cout << "Checking Correctness with Gunrock..." << endl;
    checkTCCorrectnessWithSlabGraph(device_tc_array, filenameforCorrection);
}

void triangleCountSortedVertexCentric(ll totalvertices, ll *csr_offsets, ll *csr_edges,  string &filenameforCorrection){
    // Timing Calculations
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    float totalTime = 0.0;
    float time;

    unsigned int *device_tc_array;
    hipMalloc(&device_tc_array, (totalvertices) * sizeof(unsigned int));

    unsigned blocks = ceil((double) totalvertices / BLOCKSIZE);

    cout << endl;
    cout << "Launching Sorted Vertex Centric TC Kernel" << endl;

    // Kernel for TC
    time = 0.0;
    hipEventRecord(start);
    triangleCountSortedVertexCentricKernel<<<blocks, BLOCKSIZE>>>(csr_offsets, csr_edges, device_tc_array, totalvertices);
    hipDeviceSynchronize();

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    totalTime += time;
    hipDeviceSynchronize();

    cout << "Finished Kernel" << endl;

    cout << "Total Time: " << totalTime << endl;
    cout << "First 40 values of TC: ";
    printTC<<<1,1>>>(totalvertices, device_tc_array);
    hipDeviceSynchronize();
    cout << endl;

    cout << "Checking Correctness with Gunrock..." << endl;
    checkTCCorrectnessWithSlabGraph(device_tc_array, filenameforCorrection);
}