#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "preprocessing.h"

__global__ void generateAdjListParallel(ll vertices, ll *index, ll *headVertex, ll *weights, Node *nodequeue, Node **adjList, ll *vertexToIndexMap, ll prevEdgeCount)
{
    unsigned int u = blockIdx.x * blockDim.x + threadIdx.x;

    if (u >= vertices)
        return;
    // printf("%ld\n", u);

    ll u_data = vertexToIndexMap[u];
    // printf("%ld\n", u_data);

    ll startIdx = index[u];
    ll endIdx = index[u + 1];

    for (ll idx = startIdx; idx < endIdx; ++idx)
    {
        ll v = headVertex[idx];
        ll wt = weights[idx];

        Node *node = nodequeue + prevEdgeCount + idx;
        // printf("%p\n", node);
        // if(node == NULL) printf("ye");
        // else printf("no");

        // if(node == NULL) printf("%ld ",qIndex);
        // else printf("no ");

        node->vertex = v;
        node->wt = wt;
        node->next = NULL;

        Node *temp = adjList[u_data];
        // printf("%ld ", node->vertex);

        if (!temp)
            adjList[u_data] = node;
        else
        {
            node->next = temp;
            adjList[u_data] = node;
        }
    }
}

__global__ void copyAdjacencyList(Node **oldList, Node **newList, ll oldSize)
{
    unsigned id = blockIdx.x * blockDim.x + threadIdx.x;

    if (id > oldSize)
        return;

    Node *node = oldList[id];

    newList[id] = node;
}

__global__ void updateOldToNew(Node **deviceAdjList, Node **newdeviceAdjList, Node **temp)
{
    temp = deviceAdjList;
    deviceAdjList = newdeviceAdjList;
}

__global__ void printAdjListKernel(ll vertices, Node **adjList)
{
    printf("---------------------------------STARTED PRINTING--------------------------------------\n");
    for (ll u = 0; u < vertices; ++u)
    {
        printf("%ld: ", u);

        Node *temp = adjList[u];

        // if(temp) printf("%ld ", temp->vertex);

        while (temp)
        {
            printf("(%ld, %ld), ", temp->vertex, temp->wt);
            temp = temp->next;
        }

        printf("\n");
    }
}

int main()
{
    int count = 0;
    int sortedOption;
    cout << "Do you want the edge list to be in sorted order? Enter 1 for Yes or 0 for No. ";
    cin >> sortedOption;
    // cout << sortedOption << endl;

    string file1 = "Graphs/chesapeake.mtx";
    string file2 = "Graphs/inf-luxembourg_osm.mtx";
    string file3 = "Graphs/delaunay_n17.mtx";
    string file4 = "Graphs/kron_g500-logn16.mtx";
    string file5 = "Graphs/rgg_n_2_16_s0.mtx";

    string mtxFilePath = file5;
    double totalTime = 0.0;

    ifstream file(mtxFilePath);

    if (!file.is_open())
    {
        cerr << "Failed to open the file." << endl;
        return 0;
    }

    ll totalVertices;
    ll temp1, totalEdges; // for skipping the first line vertices, edges
    vector<Edge> edgeList;
    string line;
    ll batchSize = 100000;
    bool skipLineOne = true;
    ll prevEdgeCount = 0;

    // Keep count of vertices. Track Max Vertex
    map<ll, ll> vertexCount;
    ll maxVertex = 0;

    // Defining Edge queue
    Node *nodeQueue;

    // Defining deviceAdjList;
    Node **deviceAdjList;
    ll sizeOfAdjList = 0;

    int batch = 1;
    vector<double> timings;

    while (getline(file, line))
    {

        // Skip comments
        if (line[0] == '%')
            continue;

        // Skip the first line after comments
        if (skipLineOne)
        {
            istringstream iss(line);
            iss >> totalVertices >> temp1 >> totalEdges;
            skipLineOne = false;

            // Creating edge queue
            totalEdges = DIRECTED ? totalEdges : 2 * totalEdges;
            hipMalloc((Node **)&nodeQueue, totalEdges * sizeof(Node));

            // Allocating space for adjacency list on device
            hipMalloc(&deviceAdjList, totalVertices * sizeof(Node *));
            continue;
        }

        ll src, dest, wt;

        istringstream iss(line);
        if (WEIGHTED)
            iss >> src >> dest >> wt;
        else
            iss >> src >> dest;

        Edge e;
        e.src = src - 1;
        e.dest = dest - 1;
        e.wt = WEIGHTED ? wt : 1;

        edgeList.emplace_back(e);

        ++vertexCount[e.src];
        maxVertex = max(maxVertex, e.src);

        if (!DIRECTED)
        {
            e.src = dest - 1;
            e.dest = src - 1;
            e.wt = WEIGHTED ? wt : 1;

            ++vertexCount[e.src];
            maxVertex = max(maxVertex, e.src);

            edgeList.emplace_back(e);
        }

        if (edgeList.size() >= batchSize)
        {
            if (sortedOption)
                sort(edgeList.begin(), edgeList.end(), comp_Edges_and_dest);
            else
                sort(edgeList.begin(), edgeList.end(), comp_Edges);

            // printEdgeList(edgeList);
            // ++count;
            // if(count == 2) break;

            ll noOfedges = edgeList.size();
            ll vertices = vertexCount.size();
            // ++count;
            // if(count == 3) {cout << vertices << endl; break;}

            ll *hvertexToIndexMap = (ll *)malloc((vertices) * sizeof(ll));
            ll *hindex = (ll *)malloc((vertices + 1) * sizeof(ll));
            ll *hheadVertex = (ll *)malloc(noOfedges * sizeof(ll));
            ll *hweights = (ll *)malloc(noOfedges * sizeof(ll));

            constructCSR(vertices, hindex, hheadVertex, hweights, DIRECTED, WEIGHTED, edgeList, vertexCount, hvertexToIndexMap);
            // printCSR(vertices, hindex, hheadVertex, hweights, noOfedges, hvertexToIndexMap);
            // ++count;
            // if(count == 3) break;

            // Copying CSR on GPU
            ll *dvertexToIndexMap;
            ll *dindex;
            ll *dheadVertex;
            ll *dweights;

            hipMalloc(&dvertexToIndexMap, (ll)(vertices) * sizeof(ll));
            hipMalloc(&dindex, (ll)(vertices + 1) * sizeof(ll));
            hipMalloc(&dheadVertex, (ll)(noOfedges) * sizeof(ll));
            hipMalloc(&dweights, (ll)(noOfedges) * sizeof(ll));

            hipMemcpy(dvertexToIndexMap, hvertexToIndexMap, (ll)(vertices) * sizeof(ll), hipMemcpyHostToDevice);
            hipMemcpy(dindex, hindex, (ll)(vertices + 1) * sizeof(ll), hipMemcpyHostToDevice);
            hipMemcpy(dheadVertex, hheadVertex, (ll)(noOfedges) * sizeof(ll), hipMemcpyHostToDevice);
            hipMemcpy(dweights, hweights, (ll)(noOfedges) * sizeof(ll), hipMemcpyHostToDevice);

            /* Vertex Updates
            ll maxSizeNeeded = nearestPowerOf2(maxVertex);
            cout << maxSizeNeeded << endl;

            if (sizeOfAdjList == 0)
            {
                hipMalloc(&deviceAdjList, maxSizeNeeded * sizeof(Node *));
                sizeOfAdjList = maxSizeNeeded;
            }
            else if (maxVertex > sizeOfAdjList)
            {
                Node **newdeviceAdjList;
                hipMalloc(&newdeviceAdjList, maxSizeNeeded * sizeof(Node *));

                // copy old adjacency to new
                unsigned blocks = ceil((float)sizeOfAdjList / BLOCKSIZE);
                copyAdjacencyList<<<blocks, BLOCKSIZE>>>(deviceAdjList, newdeviceAdjList, sizeOfAdjList);
                hipDeviceSynchronize();
                cout << "here1" << endl;
                sizeOfAdjList = maxSizeNeeded;
                Node **temp;
                updateOldToNew<<<1, 1>>>(deviceAdjList, newdeviceAdjList, temp);
                // printAdjListKernel<<<1, 1>>>(totalVertices, deviceAdjList);
                // hipDeviceSynchronize();
                hipFree(temp);
            }
            */

            unsigned nblocks = ceil((float)vertices / BLOCKSIZE);
            // cout << nblocks <<endl; break;

            clock_t start, end;
            start = clock();
            generateAdjListParallel<<<nblocks, BLOCKSIZE>>>(vertices, dindex, dheadVertex, dweights, nodeQueue, deviceAdjList, dvertexToIndexMap, prevEdgeCount);
            hipDeviceSynchronize();
            end = clock();
            double elapsedTime = (double)(end - start) / CLOCKS_PER_SEC * 1000.0; // Convert to milliseconds
            totalTime += elapsedTime;

            timings.emplace_back(elapsedTime);

            // printAdjListKernel<<<1, 1>>>(totalVertices, deviceAdjList);
            // hipDeviceSynchronize();
            // ++count;
            // if(count == 3) break;

            // cout << "Time taken for Batch " << batch << ": " << elapsedTime << " ms" << endl;

            ++batch;
            prevEdgeCount += noOfedges;
            edgeList.clear();
            vertexCount.clear();
            hipFree(dvertexToIndexMap);
            hipFree(dindex);
            hipFree(dheadVertex);
            hipFree(dweights);
            free(hvertexToIndexMap);
            free(hindex);
            free(hheadVertex);
            free(hweights);
        }
    }

    if (edgeList.size() > 0)
    {
        if (sortedOption)
            sort(edgeList.begin(), edgeList.end(), comp_Edges_and_dest);
        else
            sort(edgeList.begin(), edgeList.end(), comp_Edges);

        ll noOfedges = edgeList.size();
        ll vertices = vertexCount.size();

        ll *hvertexToIndexMap = (ll *)malloc((vertices) * sizeof(ll));
        ll *hindex = (ll *)malloc((vertices + 1) * sizeof(ll));
        ll *hheadVertex = (ll *)malloc(noOfedges * sizeof(ll));
        ll *hweights = (ll *)malloc(noOfedges * sizeof(ll));

        constructCSR(vertices, hindex, hheadVertex, hweights, DIRECTED, WEIGHTED, edgeList, vertexCount, hvertexToIndexMap);
        // printCSR(vertices, hindex, hheadVertex, hweights, noOfedges, hvertexToIndexMap);

        // Copying CSR on GPU
        ll *dvertexToIndexMap;
        ll *dindex;
        ll *dheadVertex;
        ll *dweights;

        hipMalloc(&dvertexToIndexMap, (ll)(vertices) * sizeof(ll));
        hipMalloc(&dindex, (ll)(vertices + 1) * sizeof(ll));
        hipMalloc(&dheadVertex, (ll)(noOfedges) * sizeof(ll));
        hipMalloc(&dweights, (ll)(noOfedges) * sizeof(ll));

        hipMemcpy(dvertexToIndexMap, hvertexToIndexMap, (ll)(vertices) * sizeof(ll), hipMemcpyHostToDevice);
        hipMemcpy(dindex, hindex, (ll)(vertices + 1) * sizeof(ll), hipMemcpyHostToDevice);
        hipMemcpy(dheadVertex, hheadVertex, (ll)(noOfedges) * sizeof(ll), hipMemcpyHostToDevice);
        hipMemcpy(dweights, hweights, (ll)(noOfedges) * sizeof(ll), hipMemcpyHostToDevice);

        /* Vertex Updates
        ll maxSizeNeeded = nearestPowerOf2(maxVertex);
        cout << maxSizeNeeded << endl;

        if (sizeOfAdjList == 0)
        {
            hipMalloc(&deviceAdjList, maxSizeNeeded * sizeof(Node *));
            sizeOfAdjList = maxSizeNeeded;
        }
        else if (maxVertex > sizeOfAdjList)
        {
            Node **newdeviceAdjList;
            hipMalloc(&newdeviceAdjList, maxSizeNeeded * sizeof(Node *));

            // copy old adjacency to new
            unsigned blocks = ceil((float)sizeOfAdjList / BLOCKSIZE);
            copyAdjacencyList<<<blocks, BLOCKSIZE>>>(deviceAdjList, newdeviceAdjList, sizeOfAdjList);
            hipDeviceSynchronize();
            cout << "here2" << endl;
            sizeOfAdjList = maxSizeNeeded;
            Node **temp;
            updateOldToNew<<<1, 1>>>(deviceAdjList, newdeviceAdjList, temp);
            // printAdjListKernel<<<1,1>>>(totalVertices, deviceAdjList);
            // hipDeviceSynchronize();
            hipFree(temp);
        }
        */

        unsigned nblocks = ceil((float)vertices / BLOCKSIZE);

        clock_t start, end;
        start = clock();
        generateAdjListParallel<<<nblocks, BLOCKSIZE>>>(vertices, dindex, dheadVertex, dweights, nodeQueue, deviceAdjList, dvertexToIndexMap, prevEdgeCount);
        hipDeviceSynchronize();
        end = clock();
        double elapsedTime = (double)(end - start) / CLOCKS_PER_SEC * 1000.0; // Convert to milliseconds
        totalTime += elapsedTime;
        timings.emplace_back(elapsedTime);
        // printAdjListKernel<<<1,1>>>(totalVertices, deviceAdjList);
        // hipDeviceSynchronize();

        // cout << "Time taken for Batch " << batch << ": " << elapsedTime << " ms" << endl;

        edgeList.clear();
        vertexCount.clear();
        hipFree(dvertexToIndexMap);
        hipFree(dindex);
        hipFree(dheadVertex);
        hipFree(dweights);
        free(hvertexToIndexMap);
        free(hindex);
        free(hheadVertex);
        free(hweights);
    }

    file.close();

    // printAdjListKernel<<<1, 1>>>(totalVertices, deviceAdjList);
    hipDeviceSynchronize();

    printTimings(timings);

    double avgTime = (double) totalTime / batch;
    cout << "Total Time is: " << totalTime << endl;
    cout << "Average Time is: " << avgTime << endl;

    return 0;
}