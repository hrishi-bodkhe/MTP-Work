#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "preprocessing.h"

__global__ void generateAdjListParallel(ll vertices, ll *index, ll *headVertex, ll *weights, Node *nodequeue, Node **adjList, ll *vertexToIndexMap, ll prevEdgeCount)
{
    unsigned int u = blockIdx.x * blockDim.x + threadIdx.x;

    if (u >= vertices)
        return;
    // printf("%ld\n", u);

    ll u_data = vertexToIndexMap[u];
    // printf("%ld\n", u_data);

    ll startIdx = index[u];
    ll endIdx = index[u + 1];

    for (ll idx = startIdx; idx < endIdx; ++idx)
    {
        ll v = headVertex[idx];
        ll wt = weights[idx];

        Node *node = nodequeue + prevEdgeCount + idx;
        // printf("%p\n", node);
        // if(node == NULL) printf("ye");
        // else printf("no");

        // if(node == NULL) printf("%ld ",qIndex);
        // else printf("no ");

        node->vertex = v;
        node->wt = wt;
        node->next = NULL;

        Node *temp = adjList[u_data];
        // printf("%ld ", node->vertex);

        if (!temp)
            adjList[u_data] = node;
        else
        {
            node->next = temp;
            adjList[u_data] = node;
        }
    }
}

__global__ void printAdjListKernel(ll vertices, Node **adjList)
{
    printf("---------------------------------STARTED PRINTING--------------------------------------\n");
    for (ll u = 0; u < vertices; ++u)
    {
        printf("%ld: ", u);

        Node *temp = adjList[u];

        // if(temp) printf("%ld ", temp->vertex);

        while (temp)
        {
            printf("(%ld, %ld), ", temp->vertex, temp->wt);
            temp = temp->next;
        }

        printf("\n");
    }
}

int main()
{
    int count = 0;
    int sortedOption;
    cout << "Do you want the edge list to be in sorted order? Enter 1 for Yes or 0 for No. ";
    cin >> sortedOption;
    // cout << sortedOption << endl;

    string file1 = "inf-luxembourg_osm.mtx";
    string file2 = "chesapeake.mtx";
    string file3 = "delaunay_n17.mtx";
    string file4 = "kron_g500-logn16.mtx";
    string file5 = "rgg_n_2_16_s0.mtx";
    string file6 = "kron_g500-logn21.mtx";

    string mtxFilePath = file1;

    ifstream file(mtxFilePath);

    if (!file.is_open())
    {
        cerr << "Failed to open the file." << endl;
        return 0;
    }

    ll totalVertices;
    ll temp1, totalEdges; // for skipping the first line vertices, edges
    vector<Edge> edgeList;
    string line;
    ll batchSize = 10;
    bool skipLineOne = true;
    ll prevEdgeCount = 0;

    // Keep count of vertices. Track Max Vertex
    map<ll, ll> vertexCount;
    ll maxVertex = 0;

    // Defining Edge queue
    Node *nodeQueue;

    // Defining deviceAdjList;
    Node **deviceAdjList;

    while (getline(file, line))
    {

        // Skip comments
        if (line[0] == '%')
            continue;

        // Skip the first line after comments
        if (skipLineOne)
        {
            istringstream iss(line);
            iss >> totalVertices >> temp1 >> totalEdges;
            skipLineOne = false;

            // Creating edge queue
            totalEdges = DIRECTED ? totalEdges : 2 * totalEdges;
            hipMalloc((Node **)&nodeQueue, totalEdges * sizeof(Node));

            // Allocating space for adjacency list on device
            hipMalloc(&deviceAdjList, totalVertices * sizeof(Node *));
            continue;
        }

        ll src, dest, wt;

        istringstream iss(line);
        if (WEIGHTED)
            iss >> src >> dest >> wt;
        else
            iss >> src >> dest;

        Edge e;
        e.src = src - 1;
        e.dest = dest - 1;
        e.wt = WEIGHTED ? wt : 1;

        edgeList.emplace_back(e);

        ++vertexCount[e.src];
        maxVertex = max(maxVertex, e.src);

        if (!DIRECTED)
        {
            e.src = dest - 1;
            e.dest = src - 1;
            e.wt = WEIGHTED ? wt : 1;

            ++vertexCount[e.src];
            maxVertex = max(maxVertex, e.src);

            edgeList.emplace_back(e);
        }

        if (edgeList.size() >= batchSize)
        {
            if (sortedOption)
                sort(edgeList.begin(), edgeList.end(), comp_Edges_and_dest);
            else
                sort(edgeList.begin(), edgeList.end(), comp_Edges);

            // printEdgeList(edgeList);
            // ++count;
            // if(count == 2) break;

            ll noOfedges = edgeList.size();
            ll vertices = vertexCount.size();
            // ++count;
            // if(count == 3) {cout << vertices << endl; break;}

            ll *hvertexToIndexMap = (ll *)malloc((vertices) * sizeof(ll));
            ll *hindex = (ll *)malloc((vertices + 1) * sizeof(ll));
            ll *hheadVertex = (ll *)malloc(noOfedges * sizeof(ll));
            ll *hweights = (ll *)malloc(noOfedges * sizeof(ll));

            constructCSR(vertices, hindex, hheadVertex, hweights, DIRECTED, WEIGHTED, edgeList, vertexCount, hvertexToIndexMap);
            // printCSR(vertices, hindex, hheadVertex, hweights, noOfedges, hvertexToIndexMap);
            // ++count;
            // if(count == 3) break;

            // Copying CSR on GPU
            ll *dvertexToIndexMap;
            ll *dindex;
            ll *dheadVertex;
            ll *dweights;

            hipMalloc(&dvertexToIndexMap, (ll)(vertices) * sizeof(ll));
            hipMalloc(&dindex, (ll)(vertices + 1) * sizeof(ll));
            hipMalloc(&dheadVertex, (ll)(noOfedges) * sizeof(ll));
            hipMalloc(&dweights, (ll)(noOfedges) * sizeof(ll));

            hipMemcpy(dvertexToIndexMap, hvertexToIndexMap, (ll)(vertices) * sizeof(ll), hipMemcpyHostToDevice);
            hipMemcpy(dindex, hindex, (ll)(vertices + 1) * sizeof(ll), hipMemcpyHostToDevice);
            hipMemcpy(dheadVertex, hheadVertex, (ll)(noOfedges) * sizeof(ll), hipMemcpyHostToDevice);
            hipMemcpy(dweights, hweights, (ll)(noOfedges) * sizeof(ll), hipMemcpyHostToDevice);

            unsigned nblocks = ceil((float)vertices / BLOCKSIZE);
            // cout << nblocks <<endl; break;

            clock_t start, end;
            start = clock();
            generateAdjListParallel<<<nblocks, BLOCKSIZE>>>(vertices, dindex, dheadVertex, dweights, nodeQueue, deviceAdjList, dvertexToIndexMap, prevEdgeCount);
            hipDeviceSynchronize();
            end = clock();
            double elapsedTime = (double)(end - start) / CLOCKS_PER_SEC * 1000.0; // Convert to milliseconds

            // printAdjListKernel<<<1, 1>>>(totalVertices, deviceAdjList);
            // hipDeviceSynchronize();
            // ++count;
            // if(count == 3) break;

            // cout << "Time taken is: " << elapsedTime << " ms" << endl;
            prevEdgeCount += noOfedges;
            edgeList.clear();
            vertexCount.clear();
            hipFree(dvertexToIndexMap);
            hipFree(dindex);
            hipFree(dheadVertex);
            hipFree(dweights);
            free(hvertexToIndexMap);
            free(hindex);
            free(hheadVertex);
            free(hweights);
        }
    }

    if (edgeList.size() > 0)
    {
        if (sortedOption)
            sort(edgeList.begin(), edgeList.end(), comp_Edges_and_dest);
        else
            sort(edgeList.begin(), edgeList.end(), comp_Edges);

        ll noOfedges = edgeList.size();
        ll vertices = vertexCount.size();

        ll *hvertexToIndexMap = (ll *)malloc((vertices) * sizeof(ll));
        ll *hindex = (ll *)malloc((vertices + 1) * sizeof(ll));
        ll *hheadVertex = (ll *)malloc(noOfedges * sizeof(ll));
        ll *hweights = (ll *)malloc(noOfedges * sizeof(ll));

        constructCSR(vertices, hindex, hheadVertex, hweights, DIRECTED, WEIGHTED, edgeList, vertexCount, hvertexToIndexMap);
        // printCSR(vertices, hindex, hheadVertex, hweights, noOfedges, hvertexToIndexMap);

        // Copying CSR on GPU
        ll *dvertexToIndexMap;
        ll *dindex;
        ll *dheadVertex;
        ll *dweights;

        hipMalloc(&dvertexToIndexMap, (ll)(vertices) * sizeof(ll));
        hipMalloc(&dindex, (ll)(vertices + 1) * sizeof(ll));
        hipMalloc(&dheadVertex, (ll)(noOfedges) * sizeof(ll));
        hipMalloc(&dweights, (ll)(noOfedges) * sizeof(ll));

        hipMemcpy(dvertexToIndexMap, hvertexToIndexMap, (ll)(vertices) * sizeof(ll), hipMemcpyHostToDevice);
        hipMemcpy(dindex, hindex, (ll)(vertices + 1) * sizeof(ll), hipMemcpyHostToDevice);
        hipMemcpy(dheadVertex, hheadVertex, (ll)(noOfedges) * sizeof(ll), hipMemcpyHostToDevice);
        hipMemcpy(dweights, hweights, (ll)(noOfedges) * sizeof(ll), hipMemcpyHostToDevice);

        unsigned nblocks = ceil((float)vertices / BLOCKSIZE);

        clock_t start, end;
        start = clock();
        generateAdjListParallel<<<nblocks, BLOCKSIZE>>>(vertices, dindex, dheadVertex, dweights, nodeQueue, deviceAdjList, dvertexToIndexMap, prevEdgeCount);
        hipDeviceSynchronize();
        end = clock();
        double elapsedTime = (double)(end - start) / CLOCKS_PER_SEC * 1000.0; // Convert to milliseconds
        // printAdjListKernel<<<1,1>>>(totalVertices, deviceAdjList);
        hipDeviceSynchronize();

        // cout << "Time taken is: " << elapsedTime << " ms" << endl;
        edgeList.clear();
        vertexCount.clear();
        hipFree(dvertexToIndexMap);
        hipFree(dindex);
        hipFree(dheadVertex);
        hipFree(dweights);
        free(hvertexToIndexMap);
        free(hindex);
        free(hheadVertex);
        free(hweights);
    }

    file.close();

    printAdjListKernel<<<1,1>>>(totalVertices, deviceAdjList);
    hipDeviceSynchronize();
    return 0;
}